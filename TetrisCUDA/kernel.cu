#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <limits> 
#include <stdio.h>
#include <vector>
#include <string>
#define NOMINMAX
#include <Windows.h>
#include <random>
#include <cmath>
#include <map>
#include "hip/hip_runtime.h"
#include ""

std::random_device rd; // Инициализация генератора случайных чисел
#define ID_NEW_GAME_BUTTON 15001
#define ID_EXIT_BUTTON 15002
// В глобальной области видимости объявляем идентификатор таймера
#define IDT_TIMER1 1

// Структура Point для работы на устройстве и хосте
struct Point {
    int x;
    int y;

    __host__ __device__ Point(int x_ = 0, int y_ = 0) : x(x_), y(y_) {}

    __host__ __device__ void setX(int newX) { x = newX; }
    __host__ __device__ void setY(int newY) { y = newY; }
};

// Класс Tetromino для работы на устройстве и хосте
class Tetromino {
public:
    std::vector<Point> pointsRect1;
    std::vector<Point> pointsRect2;
    std::vector<Point> pointsRect3;
    std::vector<Point> pointsRect4;

    __host__ __device__ Tetromino() {}

    // Конструктор
    __host__ __device__ Tetromino(const std::vector<Point>& p1, const std::vector<Point>& p2,
        const std::vector<Point>& p3, const std::vector<Point>& p4)
        : pointsRect1(p1), pointsRect2(p2), pointsRect3(p3), pointsRect4(p4) {}

    // Методы для доступа к точкам
    __host__ __device__ const std::vector<Point>& getPointsRect1() const { return pointsRect1; }
    __host__ __device__ const std::vector<Point>& getPointsRect2() const { return pointsRect2; }
    __host__ __device__ const std::vector<Point>& getPointsRect3() const { return pointsRect3; }
    __host__ __device__ const std::vector<Point>& getPointsRect4() const { return pointsRect4; }

    // Методы для изменения точек
    __host__ __device__ void setPointsRect1(const std::vector<Point>& points) { pointsRect1 = points; }
    __host__ __device__ void setPointsRect2(const std::vector<Point>& points) { pointsRect2 = points; }
    __host__ __device__ void setPointsRect3(const std::vector<Point>& points) { pointsRect3 = points; }
    __host__ __device__ void setPointsRect4(const std::vector<Point>& points) { pointsRect4 = points; }
};

// Ядро для масштабирования точек
__global__ void ScaleKernel(int scale, Point* points, int* offsets, int numTetrominos, int totalPoints) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= totalPoints) return;

    int tetrominoIdx = 0;
    while (tetrominoIdx < numTetrominos - 1 && offsets[tetrominoIdx + 1] <= idx) {
        ++tetrominoIdx;
    }

    points[idx].x *= scale;
    points[idx].y *= scale;
}

// Основная функция для масштабирования
void Scale(int scale, std::vector<std::vector<Tetromino>>& Figs) {
    int numFigs = Figs.size();
    std::vector<int> offsets;
    int totalPoints = 0;

    for (const auto& tetrominoList : Figs) {
        for (const auto& tetromino : tetrominoList) {
            offsets.push_back(totalPoints);
            totalPoints += tetromino.getPointsRect1().size();
            totalPoints += tetromino.getPointsRect2().size();
            totalPoints += tetromino.getPointsRect3().size();
            totalPoints += tetromino.getPointsRect4().size();
        }
    }
    offsets.push_back(totalPoints);

    Point* d_points;
    int* d_offsets;
    hipMalloc(&d_points, totalPoints * sizeof(Point));
    hipMalloc(&d_offsets, offsets.size() * sizeof(int));
    hipMemcpy(d_offsets, offsets.data(), offsets.size() * sizeof(int), hipMemcpyHostToDevice);

    std::vector<Point> allPoints;
    for (const auto& tetrominoList : Figs) {
        for (const auto& tetromino : tetrominoList) {
            allPoints.insert(allPoints.end(), tetromino.getPointsRect1().begin(), tetromino.getPointsRect1().end());
            allPoints.insert(allPoints.end(), tetromino.getPointsRect2().begin(), tetromino.getPointsRect2().end());
            allPoints.insert(allPoints.end(), tetromino.getPointsRect3().begin(), tetromino.getPointsRect3().end());
            allPoints.insert(allPoints.end(), tetromino.getPointsRect4().begin(), tetromino.getPointsRect4().end());
        }
    }
    hipMemcpy(d_points, allPoints.data(), totalPoints * sizeof(Point), hipMemcpyHostToDevice);

    // Запускаем ядро
    int blockSize = 256;
    int numBlocks = (totalPoints + blockSize - 1) / blockSize;
    ScaleKernel << <numBlocks, blockSize >> > (scale, d_points, d_offsets, numFigs, totalPoints);
    hipDeviceSynchronize();

    // Копируем данные обратно на хост
    hipMemcpy(allPoints.data(), d_points, totalPoints * sizeof(Point), hipMemcpyDeviceToHost);

    // Обновляем исходные данные
    int pointIdx = 0;
    for (auto& tetrominoList : Figs) {
        for (auto& tetromino : tetrominoList) {
            auto size1 = tetromino.getPointsRect1().size();
            std::vector<Point> pointsRect1(allPoints.begin() + pointIdx, allPoints.begin() + pointIdx + size1);
            pointIdx += size1;

            auto size2 = tetromino.getPointsRect2().size();
            std::vector<Point> pointsRect2(allPoints.begin() + pointIdx, allPoints.begin() + pointIdx + size2);
            pointIdx += size2;

            auto size3 = tetromino.getPointsRect3().size();
            std::vector<Point> pointsRect3(allPoints.begin() + pointIdx, allPoints.begin() + pointIdx + size3);
            pointIdx += size3;

            auto size4 = tetromino.getPointsRect4().size();
            std::vector<Point> pointsRect4(allPoints.begin() + pointIdx, allPoints.begin() + pointIdx + size4);
            pointIdx += size4;

            tetromino.setPointsRect1(pointsRect1);
            tetromino.setPointsRect2(pointsRect2);
            tetromino.setPointsRect3(pointsRect3);
            tetromino.setPointsRect4(pointsRect4);
        }
    }

    // Освобождаем память на устройстве
    hipFree(d_points);
    hipFree(d_offsets);
}


static class  TetrominoFactory
{
public: static   Tetromino CreateFig1()
{
    Point p1 = Point(0, 0);
    Point p2 = Point(10, 0);
    Point p3 = Point(10, 10);
    Point p4 = Point(0, 10);

    Point p5 = Point(11, 0);
    Point p6 = Point(21, 0);
    Point p7 = Point(21, 10);
    Point p8 = Point(11, 10);

    Point p9 = Point(22, 0);
    Point p10 = Point(32, 0);
    Point p11 = Point(32, 10);
    Point p12 = Point(22, 10);

    Point p13 = Point(33, 0);
    Point p14 = Point(43, 0);
    Point p15 = Point(43, 10);
    Point p16 = Point(33, 10);

    return  Tetromino({ p1, p2, p3, p4 }, { p5, p6, p7, p8 }, { p9, p10, p11, p12 }, { p13, p14, p15, p16 });
}

public: static Tetromino CreateFig2()
{
    Point p1 = Point(0, 0);
    Point p2 = Point(10, 0);
    Point p3 = Point(10, 10);
    Point p4 = Point(0, 10);

    Point p5 = Point(11, 0);
    Point p6 = Point(21, 0);
    Point p7 = Point(21, 10);
    Point p8 = Point(11, 10);

    Point p9 = Point(22, 0);
    Point p10 = Point(32, 0);
    Point p11 = Point(32, 10);
    Point p12 = Point(22, 10);

    Point p13 = Point(22, 11);
    Point p14 = Point(32, 11);
    Point p15 = Point(32, 21);
    Point p16 = Point(22, 21);

    return  Tetromino({ p1, p2, p3, p4 }, { p5, p6, p7, p8 }, { p9, p10, p11, p12 }, { p13, p14, p15, p16 });
}

public:  static  Tetromino CreateFig3()
{
    Point p1 = Point(0, 0);
    Point p2 = Point(10, 0);
    Point p3 = Point(10, 10);
    Point p4 = Point(0, 10);

    Point p5 = Point(11, 0);
    Point p6 = Point(21, 0);
    Point p7 = Point(21, 10);
    Point p8 = Point(11, 10);

    Point p9 = Point(22, 0);
    Point p10 = Point(32, 0);
    Point p11 = Point(32, 10);
    Point p12 = Point(22, 10);

    Point p13 = Point(0, 11);
    Point p14 = Point(10, 11);
    Point p15 = Point(10, 21);
    Point p16 = Point(0, 21);

    return  Tetromino({ p1, p2, p3, p4 }, { p5, p6, p7, p8 }, { p9, p10, p11, p12 }, { p13, p14, p15, p16 });
}
public:   static    Tetromino CreateFig4()
{
    Point p1 = Point(0, 0);
    Point p2 = Point(10, 0);
    Point p3 = Point(10, 10);
    Point p4 = Point(0, 10);

    Point p5 = Point(11, 0);
    Point p6 = Point(21, 0);
    Point p7 = Point(21, 10);
    Point p8 = Point(11, 10);

    Point p9 = Point(22, 0);
    Point p10 = Point(32, 0);
    Point p11 = Point(32, 10);
    Point p12 = Point(22, 10);

    Point p13 = Point(11, 11);
    Point p14 = Point(21, 11);
    Point p15 = Point(21, 21);
    Point p16 = Point(11, 21);

    return  Tetromino({ p1, p2, p3, p4 }, { p5, p6, p7, p8 }, { p9, p10, p11, p12 }, { p13, p14, p15, p16 });
}
public:   static   Tetromino CreateFig5()
{
    Point p1 = Point(0, 0);
    Point p2 = Point(10, 0);
    Point p3 = Point(10, 10);
    Point p4 = Point(0, 10);

    Point p5 = Point(11, 0);
    Point p6 = Point(21, 0);
    Point p7 = Point(21, 10);
    Point p8 = Point(11, 10);

    Point p9 = Point(0, 11);
    Point p10 = Point(10, 11);
    Point p11 = Point(10, 21);
    Point p12 = Point(0, 21);

    Point p13 = Point(11, 11);
    Point p14 = Point(21, 11);
    Point p15 = Point(21, 21);
    Point p16 = Point(11, 21);

    return  Tetromino({ p1, p2, p3, p4 }, { p5, p6, p7, p8 }, { p9, p10, p11, p12 }, { p13, p14, p15, p16 });
}
public: static   Tetromino CreateFig6()
{
    Point p1 = Point(0, 0);
    Point p2 = Point(10, 0);
    Point p3 = Point(10, 10);
    Point p4 = Point(0, 10);

    Point p5 = Point(11, 0);
    Point p6 = Point(21, 0);
    Point p7 = Point(21, 10);
    Point p8 = Point(11, 10);

    Point p9 = Point(11, 11);
    Point p10 = Point(21, 11);
    Point p11 = Point(21, 21);
    Point p12 = Point(11, 21);

    Point p13 = Point(22, 11);
    Point p14 = Point(32, 11);
    Point p15 = Point(32, 21);
    Point p16 = Point(22, 21);

    return  Tetromino({ p1, p2, p3, p4 }, { p5, p6, p7, p8 }, { p9, p10, p11, p12 }, { p13, p14, p15, p16 });
}
public:   static  Tetromino CreateFig7()
{
    Point p1 = Point(11, 0);
    Point p2 = Point(21, 0);
    Point p3 = Point(21, 10);
    Point p4 = Point(11, 10);

    Point p5 = Point(22, 0);
    Point p6 = Point(32, 0);
    Point p7 = Point(32, 10);
    Point p8 = Point(22, 10);

    Point p9 = Point(0, 11);
    Point p10 = Point(10, 11);
    Point p11 = Point(10, 21);
    Point p12 = Point(0, 21);

    Point p13 = Point(11, 11);
    Point p14 = Point(21, 11);
    Point p15 = Point(21, 21);
    Point p16 = Point(11, 21);

    return  Tetromino({ p1, p2, p3, p4 }, { p5, p6, p7, p8 }, { p9, p10, p11, p12 }, { p13, p14, p15, p16 });
}

public:  static  Tetromino CreateFig1_1()
{
    Point p1 = Point(0, 0);
    Point p2 = Point(10, 0);
    Point p3 = Point(10, 10);
    Point p4 = Point(0, 10);

    Point p5 = Point(0, 11);
    Point p6 = Point(10, 11);
    Point p7 = Point(10, 21);
    Point p8 = Point(0, 21);

    Point p9 = Point(0, 22);
    Point p10 = Point(10, 22);
    Point p11 = Point(10, 32);
    Point p12 = Point(0, 32);

    Point p13 = Point(0, 33);
    Point p14 = Point(10, 33);
    Point p15 = Point(10, 43);
    Point p16 = Point(0, 43);

    return  Tetromino({ p1, p2, p3, p4 }, { p5, p6, p7, p8 }, { p9, p10, p11, p12 }, { p13, p14, p15, p16 });
}

public:  static   Tetromino CreateFig2_1()
{
    Point p1 = Point(11, 0);
    Point p2 = Point(21, 0);
    Point p3 = Point(21, 10);
    Point p4 = Point(11, 10);

    Point p5 = Point(11, 11);
    Point p6 = Point(21, 11);
    Point p7 = Point(21, 21);
    Point p8 = Point(11, 21);

    Point p9 = Point(11, 22);
    Point p10 = Point(21, 22);
    Point p11 = Point(21, 32);
    Point p12 = Point(11, 32);

    Point p13 = Point(0, 22);
    Point p14 = Point(10, 22);
    Point p15 = Point(10, 32);
    Point p16 = Point(0, 32);

    return  Tetromino({ p1, p2, p3, p4 }, { p5, p6, p7, p8 }, { p9, p10, p11, p12 }, { p13, p14, p15, p16 });
}


public:  static  Tetromino CreateFig2_2()
{
    Point p1 = Point(0, 11);
    Point p2 = Point(10, 11);
    Point p3 = Point(10, 21);
    Point p4 = Point(0, 21);

    Point p5 = Point(11, 11);
    Point p6 = Point(21, 11);
    Point p7 = Point(21, 21);
    Point p8 = Point(11, 21);

    Point p9 = Point(22, 11);
    Point p10 = Point(32, 11);
    Point p11 = Point(32, 21);
    Point p12 = Point(22, 21);

    Point p13 = Point(0, 0);
    Point p14 = Point(10, 0);
    Point p15 = Point(10, 10);
    Point p16 = Point(0, 10);

    return  Tetromino({ p1, p2, p3, p4 }, { p5, p6, p7, p8 }, { p9, p10, p11, p12 }, { p13, p14, p15, p16 });
}

public:  static Tetromino CreateFig2_3()
{
    Point p1 = Point(0, 0);
    Point p2 = Point(10, 0);
    Point p3 = Point(10, 10);
    Point p4 = Point(0, 10);

    Point p5 = Point(0, 11);
    Point p6 = Point(10, 11);
    Point p7 = Point(10, 21);
    Point p8 = Point(0, 21);

    Point p9 = Point(0, 22);
    Point p10 = Point(10, 22);
    Point p11 = Point(10, 32);
    Point p12 = Point(0, 32);

    Point p13 = Point(11, 0);
    Point p14 = Point(21, 0);
    Point p15 = Point(21, 10);
    Point p16 = Point(11, 10);

    return  Tetromino({ p1, p2, p3, p4 }, { p5, p6, p7, p8 }, { p9, p10, p11, p12 }, { p13, p14, p15, p16 });
}
public:  static  Tetromino CreateFig3_1()
{

    Point p1 = Point(11, 0);
    Point p2 = Point(21, 0);
    Point p3 = Point(21, 10);
    Point p4 = Point(11, 10);

    Point p5 = Point(11, 11);
    Point p6 = Point(21, 11);
    Point p7 = Point(21, 21);
    Point p8 = Point(11, 21);

    Point p9 = Point(11, 22);
    Point p10 = Point(21, 22);
    Point p11 = Point(21, 32);
    Point p12 = Point(11, 32);

    Point p13 = Point(0, 0);
    Point p14 = Point(10, 0);
    Point p15 = Point(10, 10);
    Point p16 = Point(0, 10);

    return  Tetromino({ p1, p2, p3, p4 }, { p5, p6, p7, p8 }, { p9, p10, p11, p12 }, { p13, p14, p15, p16 });
}

public: static   Tetromino CreateFig3_2()
{

    Point p1 = Point(0, 11);
    Point p2 = Point(10, 11);
    Point p3 = Point(10, 21);
    Point p4 = Point(0, 21);

    Point p5 = Point(11, 11);
    Point p6 = Point(21, 11);
    Point p7 = Point(21, 21);
    Point p8 = Point(11, 21);

    Point p9 = Point(22, 11);
    Point p10 = Point(32, 11);
    Point p11 = Point(32, 21);
    Point p12 = Point(22, 21);

    Point p13 = Point(22, 0);
    Point p14 = Point(32, 0);
    Point p15 = Point(32, 10);
    Point p16 = Point(22, 10);

    return  Tetromino({ p1, p2, p3, p4 }, { p5, p6, p7, p8 }, { p9, p10, p11, p12 }, { p13, p14, p15, p16 });
}

public: static   Tetromino CreateFig3_3()
{

    Point p1 = Point(0, 0);
    Point p2 = Point(10, 0);
    Point p3 = Point(10, 10);
    Point p4 = Point(0, 10);

    Point p5 = Point(0, 11);
    Point p6 = Point(10, 11);
    Point p7 = Point(10, 21);
    Point p8 = Point(0, 21);

    Point p9 = Point(0, 22);
    Point p10 = Point(10, 22);
    Point p11 = Point(10, 32);
    Point p12 = Point(0, 32);

    Point p13 = Point(11, 22);
    Point p14 = Point(21, 22);
    Point p15 = Point(21, 32);
    Point p16 = Point(11, 32);


    return  Tetromino({ p1, p2, p3, p4 }, { p5, p6, p7, p8 }, { p9, p10, p11, p12 }, { p13, p14, p15, p16 });
}

public: static  Tetromino CreateFig4_1()
{
    Point p1 = Point(11, 0);
    Point p2 = Point(21, 0);
    Point p3 = Point(21, 10);
    Point p4 = Point(11, 10);

    Point p5 = Point(11, 11);
    Point p6 = Point(21, 11);
    Point p7 = Point(21, 21);
    Point p8 = Point(11, 21);

    Point p9 = Point(11, 22);
    Point p10 = Point(21, 22);
    Point p11 = Point(21, 32);
    Point p12 = Point(11, 32);

    Point p13 = Point(0, 11);
    Point p14 = Point(10, 11);
    Point p15 = Point(10, 21);
    Point p16 = Point(0, 21);

    return  Tetromino({ p1, p2, p3, p4 }, { p5, p6, p7, p8 }, { p9, p10, p11, p12 }, { p13, p14, p15, p16 });
}
public:  static  Tetromino CreateFig4_2()
{
    Point p1 = Point(0, 11);
    Point p2 = Point(10, 11);
    Point p3 = Point(10, 21);
    Point p4 = Point(0, 21);

    Point p5 = Point(11, 11);
    Point p6 = Point(21, 11);
    Point p7 = Point(21, 21);
    Point p8 = Point(11, 21);

    Point p9 = Point(22, 11);
    Point p10 = Point(32, 11);
    Point p11 = Point(32, 21);
    Point p12 = Point(22, 21);

    Point p13 = Point(11, 0);
    Point p14 = Point(21, 0);
    Point p15 = Point(21, 10);
    Point p16 = Point(11, 10);


    return  Tetromino({ p1, p2, p3, p4 }, { p5, p6, p7, p8 }, { p9, p10, p11, p12 }, { p13, p14, p15, p16 });
}

public:  static   Tetromino CreateFig4_3()
{
    Point p1 = Point(0, 0);
    Point p2 = Point(10, 0);
    Point p3 = Point(10, 10);
    Point p4 = Point(0, 10);

    Point p5 = Point(0, 11);
    Point p6 = Point(10, 11);
    Point p7 = Point(10, 21);
    Point p8 = Point(0, 21);

    Point p9 = Point(0, 22);
    Point p10 = Point(10, 22);
    Point p11 = Point(10, 32);
    Point p12 = Point(0, 32);

    Point p13 = Point(11, 11);
    Point p14 = Point(21, 11);
    Point p15 = Point(21, 21);
    Point p16 = Point(11, 21);

    return  Tetromino({ p1, p2, p3, p4 }, { p5, p6, p7, p8 }, { p9, p10, p11, p12 }, { p13, p14, p15, p16 });
}

public: static   Tetromino CreateFig6_1()
{

    Point p1 = Point(11, 0);
    Point p2 = Point(21, 0);
    Point p3 = Point(21, 10);
    Point p4 = Point(11, 10);

    Point p5 = Point(11, 11);
    Point p6 = Point(21, 11);
    Point p7 = Point(21, 21);
    Point p8 = Point(11, 21);

    Point p9 = Point(0, 11);
    Point p10 = Point(10, 11);
    Point p11 = Point(10, 21);
    Point p12 = Point(0, 21);

    Point p13 = Point(0, 22);
    Point p14 = Point(10, 22);
    Point p15 = Point(10, 32);
    Point p16 = Point(0, 32);


    return  Tetromino({ p1, p2, p3, p4 }, { p5, p6, p7, p8 }, { p9, p10, p11, p12 }, { p13, p14, p15, p16 });
}

public:   static Tetromino CreateFig7_1()
{

    Point p1 = Point(0, 0);
    Point p2 = Point(10, 0);
    Point p3 = Point(10, 10);
    Point p4 = Point(0, 10);

    Point p5 = Point(11, 11);
    Point p6 = Point(21, 11);
    Point p7 = Point(21, 21);
    Point p8 = Point(11, 21);

    Point p9 = Point(0, 11);
    Point p10 = Point(10, 11);
    Point p11 = Point(10, 21);
    Point p12 = Point(0, 21);

    Point p13 = Point(11, 22);
    Point p14 = Point(21, 22);
    Point p15 = Point(21, 32);
    Point p16 = Point(11, 32);

    return  Tetromino({ p1, p2, p3, p4 }, { p5, p6, p7, p8 }, { p9, p10, p11, p12 }, { p13, p14, p15, p16 });
}

};



int height = 1023;
int width = 660;
int Score = 0;
bool StopGame = false;
int sizes[] = { 2, 4, 4, 4, 1, 2, 2 };
int scale = 3;
std::vector<Tetromino> Figrnd_out;
std::vector<Tetromino> Figrnd_group;
Tetromino Figrnd{};
int mod = 0;
int current_mod = 0;
std::vector<std::vector<Tetromino>>Figs;
int speed_moveY = 2;

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}
// Ядро CUDA для нахождения минимального значения Y для заданного X1
__global__ void minYKernel(Point* d_points, int* d_offsets, int numTetrominos, int X1, int* d_minYList) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numTetrominos) return;

    int startOffset = d_offsets[idx];
    int endOffset = d_offsets[idx + 1];
    int minY = INT_MAX;

    for (int i = startOffset; i < endOffset; ++i) {
        if (d_points[i].x == X1 && d_points[i].y < minY) {
            minY = d_points[i].y;
        }
    }
    d_minYList[idx] = minY;
}

// Основная функция для нахождения минимального Y
int minYfromOutbyX(const std::vector<Tetromino>& Figrnd_out, int X1) {
    int numTetrominos = Figrnd_out.size();
    int totalPoints = 0;

    // Подсчет общего количества точек и заполнение offset-ов
    std::vector<int> offsets(numTetrominos + 1);
    for (int i = 0; i < numTetrominos; ++i) {
        offsets[i] = totalPoints;
        totalPoints += Figrnd_out[i].getPointsRect1().size();
        totalPoints += Figrnd_out[i].getPointsRect2().size();
        totalPoints += Figrnd_out[i].getPointsRect3().size();
        totalPoints += Figrnd_out[i].getPointsRect4().size();
    }
    offsets[numTetrominos] = totalPoints;

    Point* d_points;
    int* d_offsets;
    int* d_minYList;
    hipMalloc(&d_points, totalPoints * sizeof(Point));
    hipMalloc(&d_offsets, (numTetrominos + 1) * sizeof(int));
    hipMalloc(&d_minYList, numTetrominos * sizeof(int));

    // Сбор всех точек в один массив
    std::vector<Point> allPoints;
    for (const auto& tetromino : Figrnd_out) {
        allPoints.insert(allPoints.end(), tetromino.getPointsRect1().begin(), tetromino.getPointsRect1().end());
        allPoints.insert(allPoints.end(), tetromino.getPointsRect2().begin(), tetromino.getPointsRect2().end());
        allPoints.insert(allPoints.end(), tetromino.getPointsRect3().begin(), tetromino.getPointsRect3().end());
        allPoints.insert(allPoints.end(), tetromino.getPointsRect4().begin(), tetromino.getPointsRect4().end());
    }

    hipMemcpy(d_points, allPoints.data(), totalPoints * sizeof(Point), hipMemcpyHostToDevice);
    hipMemcpy(d_offsets, offsets.data(), (numTetrominos + 1) * sizeof(int), hipMemcpyHostToDevice);

    // Запуск ядра
    int blockSize = 256;
    int numBlocks = (numTetrominos + blockSize - 1) / blockSize;
    minYKernel << <numBlocks, blockSize >> > (d_points, d_offsets, numTetrominos, X1, d_minYList);
    hipDeviceSynchronize();

    // Копирование результатов обратно на хост
    std::vector<int> minYList(numTetrominos);
    hipMemcpy(minYList.data(), d_minYList, numTetrominos * sizeof(int), hipMemcpyDeviceToHost);

    // Освобождение памяти на устройстве
    hipFree(d_points);
    hipFree(d_offsets);
    hipFree(d_minYList);

    // Возвращение минимального Y среди всех найденных
    return *std::min_element(minYList.begin(), minYList.end());
}

//
//void Scale(int scale, std::vector<std::vector<Tetromino>>& Figs)
//{
//    // Перебираем все фигуры в массиве Figs
//    for (int i = 0; i < Figs.size(); i++)
//    {
//        std::vector<Tetromino> tetrominos = Figs[i]; // Получаем массив фигур
//        int size = sizes[i]; // Получаем размер текущего массива фигур
//
//        // Перебираем каждую фигуру в массиве фигур
//        for (int j = 0; j < size; j++)
//        {
//            Tetromino& tetromino = tetrominos[j]; // Получаем фигуру по ссылке
//
//            // Создаем векторы точек перед циклом
//            std::vector<Point> Rect1, Rect2, Rect3, Rect4;
//
//            // Получаем векторы точек перед циклом
//            const std::vector<Point>& pointsRect1 = tetromino.getPointsRect1();
//            const std::vector<Point>& pointsRect2 = tetromino.getPointsRect2();
//            const std::vector<Point>& pointsRect3 = tetromino.getPointsRect3();
//            const std::vector<Point>& pointsRect4 = tetromino.getPointsRect4();
//
//            // Перебираем все точки в каждом прямоугольнике фигуры
//            for (int k = 0; k < pointsRect1.size(); k++)
//            {
//                // Масштабируем координаты каждой точки на указанный масштаб
//                Rect1.push_back(Point(pointsRect1[k].x * scale, pointsRect1[k].y * scale));
//                Rect2.push_back(Point(pointsRect2[k].x * scale, pointsRect2[k].y * scale));
//                Rect3.push_back(Point(pointsRect3[k].x * scale, pointsRect3[k].y * scale));
//                Rect4.push_back(Point(pointsRect4[k].x * scale, pointsRect4[k].y * scale));
//            }
//
//            // Устанавливаем новые векторы точек
//            tetromino.setPointsRect1(Rect1);
//            tetromino.setPointsRect2(Rect2);
//            tetromino.setPointsRect3(Rect3);
//            tetromino.setPointsRect4(Rect4);
//        }
//        Figs[i] = tetrominos;
//    }
//}
//
//int minYfromOutbyX(const std::vector<Tetromino>& Figrnd_out, int X1)
//{
//    // Создаем вектор для хранения минимальных значений Y для каждого массива точек
//    std::vector<int> minYList;
//
//    // Перебираем все элементы Figrnd_out
//    for (const Tetromino& fig : Figrnd_out)
//    {
//        // Находим минимальное значение Y для каждого массива точек в текущем элементе Figrnd_out
//        int aYt1 = std::numeric_limits<int>::max();
//        int aYt2 = std::numeric_limits<int>::max();
//        int aYt3 = std::numeric_limits<int>::max();
//        int aYt4 = std::numeric_limits<int>::max();
//
//        for (const Point& point : fig.getPointsRect1()) {
//            if (point.x == X1 && point.y < aYt1) {
//                aYt1 = point.y;
//            }
//        }
//        for (const Point& point : fig.getPointsRect2()) {
//            if (point.x == X1 && point.y < aYt2) {
//                aYt2 = point.y;
//            }
//        }
//        for (const Point& point : fig.getPointsRect3()) {
//            if (point.x == X1 && point.y < aYt3) {
//                aYt3 = point.y;
//            }
//        }
//        for (const Point& point : fig.getPointsRect4()) {
//            if (point.x == X1 && point.y < aYt4) {
//                aYt4 = point.y;
//            }
//        }
//
//        // Находим минимальное значение Y среди найденных значений
//        int minY = std::min({ aYt1, aYt2, aYt3, aYt4 });
//
//        // Добавляем найденное минимальное значение Y в вектор
//        minYList.push_back(minY);
//    }
//
//    // Находим минимальное значение Y среди всех найденных значений
//    return *std::min_element(minYList.begin(), minYList.end());
//}



bool Moving(Tetromino Figrnd, std::vector<Tetromino> Figrnd_out)
{
    auto X1 = std::min_element(Figrnd.getPointsRect1().begin(), Figrnd.getPointsRect1().end(), [](const Point& a, const Point& b) {
        return a.x < b.x;
        });
    auto Y1 = std::max_element(Figrnd.getPointsRect1().begin(), Figrnd.getPointsRect1().end(), [](const Point& a, const Point& b) {
        return a.y < b.y;
        });
    int Y1min = minYfromOutbyX(Figrnd_out, X1->x);

    auto X2 = std::min_element(Figrnd.getPointsRect2().begin(), Figrnd.getPointsRect2().end(), [](const Point& a, const Point& b) {
        return a.x < b.x;
        });
    auto Y2 = std::max_element(Figrnd.getPointsRect2().begin(), Figrnd.getPointsRect2().end(), [](const Point& a, const Point& b) {
        return a.y < b.y;
        });
    int Y2min = minYfromOutbyX(Figrnd_out, X2->x);

    auto X3 = std::min_element(Figrnd.getPointsRect3().begin(), Figrnd.getPointsRect3().end(), [](const Point& a, const Point& b) {
        return a.x < b.x;
        });
    auto Y3 = std::max_element(Figrnd.getPointsRect3().begin(), Figrnd.getPointsRect3().end(), [](const Point& a, const Point& b) {
        return a.y < b.y;
        });
    int Y3min = minYfromOutbyX(Figrnd_out, X3->x);

    auto X4 = std::min_element(Figrnd.getPointsRect4().begin(), Figrnd.getPointsRect4().end(), [](const Point& a, const Point& b) {
        return a.x < b.x;
        });
    auto Y4 = std::max_element(Figrnd.getPointsRect4().begin(), Figrnd.getPointsRect4().end(), [](const Point& a, const Point& b) {
        return a.y < b.y;
        });
    int Y4min = minYfromOutbyX(Figrnd_out, X4->x);

    if (
        Y1->y + 1 < Y1min &&
        Y2->y + 1 < Y2min &&
        Y3->y + 1 < Y3min &&
        Y4->y + 1 < Y4min
        )
    {
        return true;
    }
    return false;
}

__global__ void maxFixYKernel(Point* d_pointsRect1, Point* d_pointsRect2, Point* d_pointsRect3, Point* d_pointsRect4, int numPoints, int scale, int h, int* d_delta) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numPoints) return;

    Point* pointsRects[] = { d_pointsRect1, d_pointsRect2, d_pointsRect3, d_pointsRect4 };
    int maxY = 0;

    for (int i = 0; i < 4; ++i) {
        Point* pointsRect = pointsRects[i];
        for (int j = 0; j < numPoints; ++j) {
            int y = pointsRect[j].y;
            if (y > maxY) {
                maxY = y;
            }
        }
    }

    maxY -= 10 * scale;

    int delta = h;

    for (int i = 0; i <= h / (11 * scale); ++i) {
        int d = maxY - i * 11 * scale;
        if (abs(d) < abs(delta)) {
            delta = d;
        }
    }

    d_delta[idx] = delta;
}

int maxFixY(const Tetromino& Figrnd,  int h) {
    std::vector<Point> pointsRect1 = Figrnd.getPointsRect1();
    std::vector<Point> pointsRect2 = Figrnd.getPointsRect2();
    std::vector<Point> pointsRect3 = Figrnd.getPointsRect3();
    std::vector<Point> pointsRect4 = Figrnd.getPointsRect4();
    int numPoints = pointsRect1.size() + pointsRect2.size() + pointsRect3.size() + pointsRect4.size();

    Point* d_pointsRect1;
    Point* d_pointsRect2;
    Point* d_pointsRect3;
    Point* d_pointsRect4;
    hipMalloc(&d_pointsRect1, pointsRect1.size() * sizeof(Point));
    hipMalloc(&d_pointsRect2, pointsRect2.size() * sizeof(Point));
    hipMalloc(&d_pointsRect3, pointsRect3.size() * sizeof(Point));
    hipMalloc(&d_pointsRect4, pointsRect4.size() * sizeof(Point));
    hipMemcpy(d_pointsRect1, pointsRect1.data(), pointsRect1.size() * sizeof(Point), hipMemcpyHostToDevice);
    hipMemcpy(d_pointsRect2, pointsRect2.data(), pointsRect2.size() * sizeof(Point), hipMemcpyHostToDevice);
    hipMemcpy(d_pointsRect3, pointsRect3.data(), pointsRect3.size() * sizeof(Point), hipMemcpyHostToDevice);
    hipMemcpy(d_pointsRect4, pointsRect4.data(), pointsRect4.size() * sizeof(Point), hipMemcpyHostToDevice);

    int* d_delta;
    hipMalloc(&d_delta, numPoints * sizeof(int));

    int blockSize = 256;
    int numBlocks = (numPoints + blockSize - 1) / blockSize;
    maxFixYKernel << <numBlocks, blockSize >> > (d_pointsRect1, d_pointsRect2, d_pointsRect3, d_pointsRect4, numPoints, scale, h, d_delta);
    hipDeviceSynchronize();

    std::vector<int> delta(numPoints);
    hipMemcpy(delta.data(), d_delta, numPoints * sizeof(int), hipMemcpyDeviceToHost);

    int minDelta = *std::min_element(delta.begin(), delta.end());

    hipFree(d_pointsRect1);
    hipFree(d_pointsRect2);
    hipFree(d_pointsRect3);
    hipFree(d_pointsRect4);
    hipFree(d_delta);

    return minDelta;
}

//
//int maxFixY(const Tetromino& Figrnd, int h)
//{
//    int maxY = 0; // Инициализируем переменную для хранения максимального значения Y
//
//    // Перебираем все массивы точек в объекте Tetromino
//    for (const auto& pointsRect : { Figrnd.getPointsRect1(), Figrnd.getPointsRect2(), Figrnd.getPointsRect3(), Figrnd.getPointsRect4() })
//    {
//        // Перебираем все точки в текущем массиве точек
//        for (const auto& point : pointsRect)
//        {
//            // Обновляем значение maxY, если значение Y текущей точки больше текущего максимального значения maxY
//            if (point.y > maxY)
//            {
//                maxY = point.y;
//            }
//        }
//    }
//
//    maxY = maxY - 10 * scale;
//
//    int delta = h;
//
//    for (int i = 0; i <= h / (11 * scale); i++)
//    {
//        int d = maxY - i * 11 * scale;
//        if (std::abs(d) < std::abs(delta)) { delta = d; }
//    }
//
//    return delta;
//}




static RECT GetRECT(const std::vector<Point>& points)
{
    RECT rect;
    if (points.size() >= 4) // Проверяем, содержит ли вектор как минимум четыре точки
    {
        rect.left = points[0].x;
        rect.top = points[0].y;
        rect.right = points[2].x;
        rect.bottom = points[2].y;
    }
    else // Если вектор содержит меньше четырех точек, используем значения по умолчанию
    {
        rect.left = 0;
        rect.top = 0;
        rect.right = 0;
        rect.bottom = 0;
    }
    return rect;
}

void DrawTetrominoRects(HDC hdc, Tetromino& tetromino, COLORREF color) {
    HBRUSH hBrush = CreateSolidBrush(color); // Создаем кисть с нужным цветом
    RECT rect1 = GetRECT(tetromino.getPointsRect1());
    FillRect(hdc, &rect1, hBrush);
    RECT rect2 = GetRECT(tetromino.getPointsRect2());
    FillRect(hdc, &rect2, hBrush);
    RECT rect3 = GetRECT(tetromino.getPointsRect3());
    FillRect(hdc, &rect3, hBrush);
    RECT rect4 = GetRECT(tetromino.getPointsRect4());
    FillRect(hdc, &rect4, hBrush);

    //FillRect(hdc, &GetRECT(tetromino.getPointsRect1()), hBrush);
    //FillRect(hdc, &GetRECT(tetromino.getPointsRect2()), hBrush);
    //FillRect(hdc, &GetRECT(tetromino.getPointsRect3()), hBrush);
    //FillRect(hdc, &GetRECT(tetromino.getPointsRect4()), hBrush);
    DeleteObject(hBrush); // Освобождаем ресурсы кисти
}

__global__ void moveXYKernel(Point* d_pointsRect1, Point* d_pointsRect2, Point* d_pointsRect3, Point* d_pointsRect4, int numPoints, int deltaX, int deltaY) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numPoints) return;

    d_pointsRect1[idx].x += deltaX;
    d_pointsRect1[idx].y += deltaY;

    d_pointsRect2[idx].x += deltaX;
    d_pointsRect2[idx].y += deltaY;

    d_pointsRect3[idx].x += deltaX;
    d_pointsRect3[idx].y += deltaY;

    d_pointsRect4[idx].x += deltaX;
    d_pointsRect4[idx].y += deltaY;
}

void MoveXY(Tetromino& t, int deltaX, int deltaY) {
    std::vector<Point> pointsRect1 = t.getPointsRect1();
    std::vector<Point> pointsRect2 = t.getPointsRect2();
    std::vector<Point> pointsRect3 = t.getPointsRect3();
    std::vector<Point> pointsRect4 = t.getPointsRect4();
    int numPoints = pointsRect1.size();

    Point* d_pointsRect1;
    Point* d_pointsRect2;
    Point* d_pointsRect3;
    Point* d_pointsRect4;
    hipMalloc(&d_pointsRect1, numPoints * sizeof(Point));
    hipMalloc(&d_pointsRect2, numPoints * sizeof(Point));
    hipMalloc(&d_pointsRect3, numPoints * sizeof(Point));
    hipMalloc(&d_pointsRect4, numPoints * sizeof(Point));
    hipMemcpy(d_pointsRect1, pointsRect1.data(), numPoints * sizeof(Point), hipMemcpyHostToDevice);
    hipMemcpy(d_pointsRect2, pointsRect2.data(), numPoints * sizeof(Point), hipMemcpyHostToDevice);
    hipMemcpy(d_pointsRect3, pointsRect3.data(), numPoints * sizeof(Point), hipMemcpyHostToDevice);
    hipMemcpy(d_pointsRect4, pointsRect4.data(), numPoints * sizeof(Point), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (numPoints + blockSize - 1) / blockSize;
    moveXYKernel << <numBlocks, blockSize >> > (d_pointsRect1, d_pointsRect2, d_pointsRect3, d_pointsRect4, numPoints, deltaX, deltaY);
    hipDeviceSynchronize();

    hipMemcpy(pointsRect1.data(), d_pointsRect1, numPoints * sizeof(Point), hipMemcpyDeviceToHost);
    hipMemcpy(pointsRect2.data(), d_pointsRect2, numPoints * sizeof(Point), hipMemcpyDeviceToHost);
    hipMemcpy(pointsRect3.data(), d_pointsRect3, numPoints * sizeof(Point), hipMemcpyDeviceToHost);
    hipMemcpy(pointsRect4.data(), d_pointsRect4, numPoints * sizeof(Point), hipMemcpyDeviceToHost);

    t.setPointsRect1(pointsRect1);
    t.setPointsRect2(pointsRect2);
    t.setPointsRect3(pointsRect3);
    t.setPointsRect4(pointsRect4);

    hipFree(d_pointsRect1);
    hipFree(d_pointsRect2);
    hipFree(d_pointsRect3);
    hipFree(d_pointsRect4);
}


//
//void MoveXY(Tetromino& t, int deltaX, int deltaY)
//{
//
//    // Создаем векторы точек перед циклом
//    std::vector<Point> Rect1, Rect2, Rect3, Rect4;
//
//    // Получаем векторы точек перед циклом
//    const std::vector<Point>& pointsRect1 = t.getPointsRect1();
//    const std::vector<Point>& pointsRect2 = t.getPointsRect2();
//    const std::vector<Point>& pointsRect3 = t.getPointsRect3();
//    const std::vector<Point>& pointsRect4 = t.getPointsRect4();
//
//    for (int k = 0; k < pointsRect1.size(); k++)
//    {
//        // Масштабируем координаты каждой точки на указанный масштаб
//        Rect1.push_back(Point(pointsRect1[k].x + deltaX, pointsRect1[k].y + deltaY));
//        Rect2.push_back(Point(pointsRect2[k].x + deltaX, pointsRect2[k].y + deltaY));
//        Rect3.push_back(Point(pointsRect3[k].x + deltaX, pointsRect3[k].y + deltaY));
//        Rect4.push_back(Point(pointsRect4[k].x + deltaX, pointsRect4[k].y + deltaY));
//    }
//
//    // Устанавливаем новые векторы точек
//    t.setPointsRect1(Rect1);
//    t.setPointsRect2(Rect2);
//    t.setPointsRect3(Rect3);
//    t.setPointsRect4(Rect4);
//}

__global__ void updateRectPointsKernel(Point* pointsRect, int numPoints, int dY) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numPoints) return;

    pointsRect[idx].y -= dY;
}

void updateRectPoints(Tetromino& fig, int dY) {
    std::vector<Point> pointsRect1 = fig.getPointsRect1();
    std::vector<Point> pointsRect2 = fig.getPointsRect2();
    std::vector<Point> pointsRect3 = fig.getPointsRect3();
    std::vector<Point> pointsRect4 = fig.getPointsRect4();
    int numPoints = pointsRect1.size();

    Point* d_pointsRect1;
    Point* d_pointsRect2;
    Point* d_pointsRect3;
    Point* d_pointsRect4;
    hipMalloc(&d_pointsRect1, numPoints * sizeof(Point));
    hipMalloc(&d_pointsRect2, numPoints * sizeof(Point));
    hipMalloc(&d_pointsRect3, numPoints * sizeof(Point));
    hipMalloc(&d_pointsRect4, numPoints * sizeof(Point));
    hipMemcpy(d_pointsRect1, pointsRect1.data(), numPoints * sizeof(Point), hipMemcpyHostToDevice);
    hipMemcpy(d_pointsRect2, pointsRect2.data(), numPoints * sizeof(Point), hipMemcpyHostToDevice);
    hipMemcpy(d_pointsRect3, pointsRect3.data(), numPoints * sizeof(Point), hipMemcpyHostToDevice);
    hipMemcpy(d_pointsRect4, pointsRect4.data(), numPoints * sizeof(Point), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (numPoints + blockSize - 1) / blockSize;
    updateRectPointsKernel << <numBlocks, blockSize >> > (d_pointsRect1, numPoints, dY);
    updateRectPointsKernel << <numBlocks, blockSize >> > (d_pointsRect2, numPoints, dY);
    updateRectPointsKernel << <numBlocks, blockSize >> > (d_pointsRect3, numPoints, dY);
    updateRectPointsKernel << <numBlocks, blockSize >> > (d_pointsRect4, numPoints, dY);
    hipDeviceSynchronize();

    hipMemcpy(pointsRect1.data(), d_pointsRect1, numPoints * sizeof(Point), hipMemcpyDeviceToHost);
    hipMemcpy(pointsRect2.data(), d_pointsRect2, numPoints * sizeof(Point), hipMemcpyDeviceToHost);
    hipMemcpy(pointsRect3.data(), d_pointsRect3, numPoints * sizeof(Point), hipMemcpyDeviceToHost);
    hipMemcpy(pointsRect4.data(), d_pointsRect4, numPoints * sizeof(Point), hipMemcpyDeviceToHost);

    fig.setPointsRect1(pointsRect1);
    fig.setPointsRect2(pointsRect2);
    fig.setPointsRect3(pointsRect3);
    fig.setPointsRect4(pointsRect4);

    hipFree(d_pointsRect1);
    hipFree(d_pointsRect2);
    hipFree(d_pointsRect3);
    hipFree(d_pointsRect4);
}

//void updateRectPoints(Tetromino& fig, int dY) {
//    auto pointsRect1 = fig.getPointsRect1();
//    for (auto& point : pointsRect1) {
//        point.y -= dY;
//    }
//    fig.setPointsRect1(pointsRect1);
//
//    auto pointsRect2 = fig.getPointsRect2();
//    for (auto& point : pointsRect2) {
//        point.y -= dY;
//    }
//    fig.setPointsRect2(pointsRect2);
//
//    auto pointsRect3 = fig.getPointsRect3();
//    for (auto& point : pointsRect3) {
//        point.y -= dY;
//    }
//    fig.setPointsRect3(pointsRect3);
//
//    auto pointsRect4 = fig.getPointsRect4();
//    for (auto& point : pointsRect4) {
//        point.y -= dY;
//    }
//    fig.setPointsRect4(pointsRect4);
//}
__global__ void incrementYlinesKernel(int* lines, const Point* rect, int rectSize, int scale, int height) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= rectSize) return;

    int Ymin = rect[idx].y;
    int y = Ymin / (scale * 11);

    if (Ymin < height) {
        atomicAdd(&lines[y], 1);
    }
}

void incrementYlines(std::map<int, int>& lines, const std::vector<Point>& rect) {
    if (rect.empty()) return;

    // Находим минимальное значение Y в векторе точек
    auto minPointIt = std::min_element(rect.begin(), rect.end(), [](const Point& a, const Point& b) {
        return a.y < b.y;
        });

    if (minPointIt != rect.end()) {
        int Ymin = minPointIt->y;
        int y = Ymin / (scale * 11);

        if (Ymin < height) {
            lines[y]++;
        }
    }
}

void incrementYlinesCuda(std::map<int, int>& lines, const std::vector<Point>& rect) {
    if (rect.empty()) return;

    int rectSize = rect.size();
    Point* d_rect;
    int* d_lines;
    int maxLines = height / (scale * 11) + 1;

    std::vector<int> linesVec(maxLines, 0);

    hipMalloc(&d_rect, rectSize * sizeof(Point));
    hipMalloc(&d_lines, maxLines * sizeof(int));
    hipMemcpy(d_rect, rect.data(), rectSize * sizeof(Point), hipMemcpyHostToDevice);
    hipMemcpy(d_lines, linesVec.data(), maxLines * sizeof(int), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (rectSize + blockSize - 1) / blockSize;
    incrementYlinesKernel << <numBlocks, blockSize >> > (d_lines, d_rect, rectSize, scale, height);
    hipDeviceSynchronize();

    hipMemcpy(linesVec.data(), d_lines, maxLines * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_rect);
    hipFree(d_lines);

    for (int i = 0; i < maxLines; ++i) {
        if (linesVec[i] > 0) {
            lines[i] += linesVec[i];
        }
    }
}

//void incrementYlines(std::map<int, int>& lines, const std::vector<Point>& rect) {
//    // Находим минимальное значение Y в векторе точек
//    auto minPointIt = std::min_element(rect.begin(), rect.end(), [](const Point& a, const Point& b) {
//        return a.y < b.y;
//        });
//
//    if (minPointIt != rect.end()) {
//        int Ymin = minPointIt->y;
//        int y = Ymin / (scale * 11);
//
//        if (Ymin < height) {
//            // Увеличиваем количество заполненных клеток в соответствующей строке
//            lines[y]++;
//        }
//    }
//}
__global__ void deleteLineRectKernel(Point* rect, int rectSize, int coordY, int scale) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= rectSize) return;

    // Определяем минимальное значение Y с помощью атомарной операции
    __shared__ int minY;
    if (threadIdx.x == 0) {
        minY = rect[0].y;
        for (int i = 1; i < rectSize; i++) {
            atomicMin(&minY, rect[i].y);
        }
    }
    __syncthreads();

    if (coordY == minY) {
        rect[idx].x = INT_MAX;
        rect[idx].y = INT_MAX;
    }
    else if (coordY > minY) {
        rect[idx].y += scale * 11;
    }
}

void DeleteLineRect(std::vector<Point>& rect, int coordY) {
    if (rect.empty()) return;

    int rectSize = rect.size();
    Point* d_rect;

    hipMalloc(&d_rect, rectSize * sizeof(Point));
    hipMemcpy(d_rect, rect.data(), rectSize * sizeof(Point), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (rectSize + blockSize - 1) / blockSize;
    deleteLineRectKernel << <numBlocks, blockSize >> > (d_rect, rectSize, coordY, scale);
    hipDeviceSynchronize();

    hipMemcpy(rect.data(), d_rect, rectSize * sizeof(Point), hipMemcpyDeviceToHost);
    hipFree(d_rect);
}
//void DeleteLineRect(std::vector<Point>& rect, int coordY) {
//    // Находим минимальное значение Y в векторе точек
//    auto minYIt = std::min_element(rect.begin(), rect.end(), [](const Point& a, const Point& b) {
//        return a.y < b.y;
//        });
//
//    if (minYIt == rect.end()) {
//        return; // Если вектор пустой, выходим из функции
//    }
//
//    int minY = minYIt->y;
//
//    std::vector<Point> newRect = rect; // Создаем копию вектора
//
//    if (coordY == minY) {
//        for (auto& point : newRect) {
//            point.x = std::numeric_limits<int>::max();
//            point.y = std::numeric_limits<int>::max();
//        }
//    }
//    else if (coordY > minY) {
//        for (auto& point : newRect) {
//            point.y += scale * 11;
//        }
//    }
//
//    rect = newRect; // Заменяем исходный вектор модифицированной копией
//}

void DrawButton(HDC hdc, RECT rect, const char* text)
{
    // Рисуем кнопку
    DrawEdge(hdc, &rect, EDGE_RAISED, BF_RECT);

    // Заполняем фон кнопки белым цветом
    HBRUSH hBrush = CreateSolidBrush(RGB(255, 255, 255));
    FillRect(hdc, &rect, hBrush);
    DeleteObject(hBrush);

    // Устанавливаем режим прозрачного текста
    SetBkMode(hdc, TRANSPARENT);

    // Преобразуем const char* в LPCWSTR
    wchar_t wtext[100];
    MultiByteToWideChar(CP_ACP, 0, text, -1, wtext, 100);

    // Выводим текст на кнопке
    DrawText(hdc, text, -1, &rect, DT_CENTER | DT_VCENTER | DT_SINGLELINE);
}

void GameOver(HWND hwnd)
{
    // Остановить таймер
    KillTimer(hwnd, IDT_TIMER1);

    // Создаем кнопку "New Game"
    CreateWindowEx(
        0,
        "BUTTON",  // Predefined class; Unicode assumed
        "New Game",      // Button text
        WS_TABSTOP | WS_VISIBLE | WS_CHILD | BS_DEFPUSHBUTTON,  // Styles
        100,         // x position
        200,         // y position
        150,        // Button width
        50,        // Button height
        hwnd,       // Parent window
        (HMENU)ID_NEW_GAME_BUTTON,       // Button ID
        (HINSTANCE)GetWindowLongPtr(hwnd, GWLP_HINSTANCE),
        NULL);      // Pointer not needed

    // Создаем кнопку "Exit"
    CreateWindowEx(
        0,
        "BUTTON",  // Predefined class; Unicode assumed
        "Exit",      // Button text
        WS_TABSTOP | WS_VISIBLE | WS_CHILD | BS_DEFPUSHBUTTON,  // Styles
        300,         // x position
        200,         // y position
        150,        // Button width
        50,        // Button height
        hwnd,       // Parent window
        (HMENU)ID_EXIT_BUTTON,       // Button ID
        (HINSTANCE)GetWindowLongPtr(hwnd, GWLP_HINSTANCE),
        NULL);      // Pointer not needed

    // Обновляем окно
    UpdateWindow(hwnd);
}

void DestroyButtons(HWND hwnd)
{
    // Удаляем кнопку "New Game"
    while (GetDlgItem(hwnd, ID_NEW_GAME_BUTTON)) {
        DestroyWindow(GetDlgItem(hwnd, ID_NEW_GAME_BUTTON));
    }

    // Удаляем кнопку "Exit"
    while (GetDlgItem(hwnd, ID_EXIT_BUTTON)) {
        DestroyWindow(GetDlgItem(hwnd, ID_EXIT_BUTTON));
    }

    // Обновляем окно
    InvalidateRect(hwnd, NULL, TRUE);
    UpdateWindow(hwnd);
}


void SearchLines(std::vector<Tetromino>& Figrnd_out)
{

    int Ny = height / (11 * scale) + 1;
    int Nx = width / (11 * scale);
    // Создаем словарь для хранения количества заполненных клеток в каждой строке
    std::map<int, int> lines;

    // Инициализируем словарь значениями 0
    for (int i = 0; i < Ny; ++i) {
        lines[i] = 0;
    }


    // Перебираем все элементы Figrnd_out
    for (Tetromino& fig : Figrnd_out)
    {

        incrementYlines(lines, fig.getPointsRect1());
        incrementYlines(lines, fig.getPointsRect2());
        incrementYlines(lines, fig.getPointsRect3());
        incrementYlines(lines, fig.getPointsRect4());
    }
    std::vector<int> fullLines{};
    // Проверяем, какие строки полностью заполнены
    for (auto line : lines)
    {
        // Если количество заполненных клеток в строке равно ширине поля, то строка полностью заполнена
        if (line.second == Nx)
        {
            // Добавляем номер строки в список полностью заполненных строк
            fullLines.push_back(line.first * scale * 11);
        }
    }

    if (lines[0] != 0)
    {
        StopGame = true;
    }

    int sc = 0;
    for (int coordY : fullLines)
    {
        for (Tetromino& fig : Figrnd_out) {
            std::vector<Point> points1 = fig.getPointsRect1();
            std::vector<Point> points2 = fig.getPointsRect2();
            std::vector<Point> points3 = fig.getPointsRect3();
            std::vector<Point> points4 = fig.getPointsRect4();

            DeleteLineRect(points1, coordY);
            DeleteLineRect(points2, coordY);
            DeleteLineRect(points3, coordY);
            DeleteLineRect(points4, coordY);

            fig.setPointsRect1(points1);
            fig.setPointsRect2(points2);
            fig.setPointsRect3(points3);
            fig.setPointsRect4(points4);
        }
        sc++;
    }
    if (sc == 1) { Score += 100; }
    else if (sc == 2) { Score += 300; }
    else if (sc == 3) { Score += 500; }
    else if (sc == 4) { Score += 800; }

}
__device__ int findMinXInRect(const Point* rect, int rectSize) {
    int minX = INT_MAX;
    for (int i = 0; i < rectSize; ++i) {
        if (rect[i].x < minX) {
            minX = rect[i].x;
        }
    }
    return minX;
}

__global__ void findMinXInTetrominoKernel(const Point* points, int* minX, int rectSize) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= 4) return;

    minX[idx] = findMinXInRect(points + idx * rectSize, rectSize);
}

__device__ int findMaxXInRect(const Point* rect, int rectSize) {
    int maxX = INT_MIN;
    for (int i = 0; i < rectSize; ++i) {
        if (rect[i].x > maxX) {
            maxX = rect[i].x;
        }
    }
    return maxX;
}

__global__ void findMaxXInTetrominoKernel(const Point* points, int* maxX, int rectSize) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= 4) return;

    maxX[idx] = findMaxXInRect(points + idx * rectSize, rectSize);
}

int findMinXInTetromino(const Tetromino& tetromino) {
    std::vector<std::vector<Point>> points = {
        tetromino.getPointsRect1(),
        tetromino.getPointsRect2(),
        tetromino.getPointsRect3(),
        tetromino.getPointsRect4()
    };

    int* d_minX;
    hipMalloc(&d_minX, 4 * sizeof(int));

    Point* d_points;
    hipMalloc(&d_points, 4 * points[0].size() * sizeof(Point));

    for (int i = 0; i < 4; ++i) {
        hipMemcpy(d_points + i * points[i].size(), points[i].data(), points[i].size() * sizeof(Point), hipMemcpyHostToDevice);
    }

    int blockSize = 256;
    int numBlocks = (4 + blockSize - 1) / blockSize;

    findMinXInTetrominoKernel << <numBlocks, blockSize >> > (d_points, d_minX, points[0].size());
    hipDeviceSynchronize();

    std::vector<int> minX(4);
    hipMemcpy(minX.data(), d_minX, 4 * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_minX);
    hipFree(d_points);

    return *std::min_element(minX.begin(), minX.end());
}

int findMaxXInTetromino(const Tetromino& tetromino) {
    std::vector<std::vector<Point>> points = {
        tetromino.getPointsRect1(),
        tetromino.getPointsRect2(),
        tetromino.getPointsRect3(),
        tetromino.getPointsRect4()
    };

    int* d_maxX;
    hipMalloc(&d_maxX, 4 * sizeof(int));

    Point* d_points;
    hipMalloc(&d_points, 4 * points[0].size() * sizeof(Point));

    for (int i = 0; i < 4; ++i) {
        hipMemcpy(d_points + i * points[i].size(), points[i].data(), points[i].size() * sizeof(Point), hipMemcpyHostToDevice);
    }

    int blockSize = 256;
    int numBlocks = (4 + blockSize - 1) / blockSize;

    findMaxXInTetrominoKernel << <numBlocks, blockSize >> > (d_points, d_maxX, points[0].size());
    hipDeviceSynchronize();

    std::vector<int> maxX(4);
    hipMemcpy(maxX.data(), d_maxX, 4 * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_maxX);
    hipFree(d_points);

    return *std::max_element(maxX.begin(), maxX.end());
}

//int findMinXInTetromino(const Tetromino& tetromino) {
//    int minX = std::numeric_limits<int>::max();
//
//    // Векторы точек в Tetromino
//    std::vector<std::vector<Point>> points = {
//        tetromino.getPointsRect1(),
//        tetromino.getPointsRect2(),
//        tetromino.getPointsRect3(),
//        tetromino.getPointsRect4()
//    };
//
//    for (const auto& rect : points) {
//        auto minIt = std::min_element(rect.begin(), rect.end(), [](const Point& a, const Point& b) {
//            return a.x < b.x;
//            });
//
//        if (minIt != rect.end() && minIt->x < minX) {
//            minX = minIt->x;
//        }
//    }
//
//    return minX;
//}
//
//int findMaxXInTetromino(const Tetromino& tetromino) {
//    int maxX = std::numeric_limits<int>::min();
//
//    // Векторы точек в Tetromino
//    std::vector<std::vector<Point>> points = {
//        tetromino.getPointsRect1(),
//        tetromino.getPointsRect2(),
//        tetromino.getPointsRect3(),
//        tetromino.getPointsRect4()
//    };
//
//    for (const auto& rect : points) {
//        auto maxIt = std::max_element(rect.begin(), rect.end(), [](const Point& a, const Point& b) {
//            return a.x < b.x;
//            });
//
//        if (maxIt != rect.end() && maxIt->x > maxX) {
//            maxX = maxIt->x;
//        }
//    }
//
//    return maxX;
//}

// Двойная буферизация
HBITMAP hbmMem = NULL;
HDC hdcMem = NULL;
HBITMAP hbmOld = NULL;
RECT clientRect;


// Функция для включения двойной буферизации
void SetDoubleBuffered(HWND hwnd, BOOL enabled) {
    if (enabled) {
        SetWindowLong(hwnd, GWL_STYLE, GetWindowLong(hwnd, GWL_STYLE) | WS_EX_COMPOSITED);
    }
}
void StartTimer(HWND hwnd)
{
    SetTimer(hwnd, IDT_TIMER1, 10, NULL);
}
void Startgame()
{
    // Обновляем окно

    Figrnd_out.clear();
    Score = 0;
    std::mt19937 gen(rd()); // Инициализация генератора случайных чисел Mersenne Twister
    std::uniform_int_distribution<> distr(0, 6); // Инициализация равномерного распределения в заданном диапазоне
    int fig = distr(gen);
    mod = sizes[fig];
    std::uniform_int_distribution<> distr2(0, mod - 1);
    current_mod = distr2(gen);
    Figrnd_group = Figs[fig];

    int deltaX = ((width / 2) / (11 * scale)) * 11 * scale - 11 * scale;
    for (int i = 0; i < mod; i++)
    {
        MoveXY(Figrnd_group[i], deltaX, 0);
    }

    Figrnd = Figrnd_group[current_mod];


    int Ydec = std::round(static_cast<int>(height) / (scale * 11));
    int Y0 = (int)Ydec * (scale * 11);
    //Y0 = 0;
    int stepX = scale * 11;
    for (int j = 0; j <= width / (stepX * 1); j += 4)
    {
        //  Figrnd_out.
        std::vector<Point> Rect1, Rect2, Rect3, Rect4;
        {
            Rect1 = { Point(j * stepX, Y0),  Point(j * stepX + scale * 10, Y0),  Point(j * stepX + scale * 10, Y0 + scale * 10),  Point(j * stepX, Y0 + scale * 10) };
            Rect2 = { Point((j + 1) * stepX, Y0),  Point((j + 1) * stepX + scale * 10, Y0),  Point((j + 1) * stepX + scale * 10, Y0 + scale * 10),  Point((j + 1) * stepX, Y0 + scale * 10) };
            Rect3 = { Point((j + 2) * stepX, Y0),  Point((j + 2) * stepX + scale * 10, Y0),  Point((j + 2) * stepX + scale * 10, Y0 + scale * 10),  Point((j + 2) * stepX, Y0 + scale * 10) };
            Rect4 = { Point((j + 3) * stepX, Y0),  Point((j + 3) * stepX + scale * 10, Y0),  Point((j + 3) * stepX + scale * 10, Y0 + scale * 10),  Point((j + 3) * stepX, Y0 + scale * 10) };
            Figrnd_out.push_back(Tetromino(Rect1, Rect2, Rect3, Rect4));
        }
    }
}



LRESULT CALLBACK WindowProc(HWND hwnd, UINT uMsg, WPARAM wParam, LPARAM lParam) {


    switch (uMsg) {

    case WM_COMMAND:
        switch (LOWORD(wParam))
        {
        case ID_NEW_GAME_BUTTON:
            // Обработка нажатия на кнопку "New Game"
            DestroyButtons(hwnd);

            StopGame = false;
            UpdateWindow(hwnd);
            Startgame();
            // InvalidateRect(hwnd, NULL, TRUE); // Перерисовываем окно
            StartTimer(hwnd); // Запуск таймера заново
            // Логика для новой игры...
            break;
        case ID_EXIT_BUTTON:
            // Обработка нажатия на кнопку "Exit"
            DestroyButtons(hwnd);
            PostQuitMessage(0);
            break;
        }
        break;


    case WM_KEYDOWN:
        switch (wParam) {
        case 'W':
        case VK_UP:
        {
            // MoveXY(Figrnd_group[i], 0, 1);
            int max = Figrnd_group.size();
            if (current_mod + 1 < max)
            {
                ++current_mod;
            }
            else { current_mod = 0; }
            Figrnd = Figrnd_group[current_mod];


            // InvalidateRect(hwnd, NULL, TRUE); // Перерисовываем окно
            break;
        }
        case 'S':
        case VK_DOWN:
        {
            // MoveXY(Figrnd_group[i], 0, 1);
            int max = Figrnd_group.size();
            if (current_mod - 1 >= 0)
            {
                --current_mod;
            }
            else { current_mod = max - 1; }
            Figrnd = Figrnd_group[current_mod];


            // InvalidateRect(hwnd, NULL, TRUE); // Перерисовываем окно
            break;
        }
        case 'A':
        case VK_LEFT:
        {
            int max = Figrnd_group.size();
            int min = width;
            int minx = 0;

            for (int i = 0; i < max; i++) {
                minx = findMinXInTetromino(Figrnd_group[i]);

                if (minx < min) { min = minx; }
            }
            if (min > 0) {
                for (int i = 0; i < max; i++) {
                    MoveXY(Figrnd_group[i], -scale * 11, 0);
                }
                //  InvalidateRect(hwnd, NULL, TRUE); // Перерисовываем окно
            }
            break;
        }
        case 'D':
        case VK_RIGHT:
        {
            int max = Figrnd_group.size();
            int max_ = 0;
            int maxx = 0;

            for (int i = 0; i < max; i++) {
                maxx = findMaxXInTetromino(Figrnd_group[i]);

                if (maxx > max_) { max_ = maxx; }
            }
            int w = (11 * scale) * width / (11 * scale);

            for (int i = 0; i < max; i++) {
                maxx = findMaxXInTetromino(Figrnd_group[i]);
                if (maxx + (11 * scale) <= width) {
                    MoveXY(Figrnd_group[i], scale * 11, 0);
                }
            }
            //   InvalidateRect(hwnd, NULL, TRUE); // Перерисовываем окно

            break;
        }
        case VK_SPACE:
        case VK_RETURN:
        {
            speed_moveY = 10;
            // InvalidateRect(hwnd, NULL, TRUE); // Перерисовываем окно
            break;
        }
        break;
        }



    case WM_CREATE:
    {
        SetDoubleBuffered(hwnd, true);
        SetTimer(hwnd, IDT_TIMER1, 10, NULL);
        GetClientRect(hwnd, &clientRect);
        hdcMem = CreateCompatibleDC(NULL);
        hbmMem = CreateCompatibleBitmap(GetDC(hwnd), clientRect.right - clientRect.left, clientRect.bottom - clientRect.top);
        hbmOld = (HBITMAP)SelectObject(hdcMem, hbmMem);
        break;
    }

    case WM_TIMER:
    {
        if (StopGame) {

            KillTimer(hwnd, IDT_TIMER1);
            break;
        }
        else {}

        // Проверяем, что сообщение пришло от нашего таймера
        if (wParam == IDT_TIMER1)
        {
            Figrnd = Figrnd_group[current_mod];

            for (int i = 0; i < mod; i++)
            {
                if (
                    Moving(Figrnd, Figrnd_out)
                    )
                {

                    MoveXY(Figrnd_group[i], 0, speed_moveY);
                }
                else
                {
                    speed_moveY = 2;
                    int dY = maxFixY(Figrnd, height);
                    // выравнивание позиций Y

                    updateRectPoints(Figrnd, dY);


                    Figrnd_out.push_back(Figrnd);

                    SearchLines(Figrnd_out);



                    std::mt19937 gen(rd()); // Инициализация генератора случайных чисел Mersenne Twister
                    std::uniform_int_distribution<> distr(0, 6); // Инициализация равномерного распределения в заданном диапазоне
                    int fig = distr(gen);
                    mod = sizes[fig];
                    std::uniform_int_distribution<> distr2(0, mod - 1);
                    current_mod = distr2(gen);
                    Figrnd_group = Figs[fig];


                    int deltaX = ((width / 2) / (11 * scale)) * 11 * scale - 11 * scale;
                    for (int i = 0; i < mod; i++)
                    {
                        MoveXY(Figrnd_group[i], deltaX, 0);
                    }

                    Figrnd = Figrnd_group[current_mod];
                }
            }


            // Вызываем функцию для перерисовки всего окна
            InvalidateRect(hwnd, NULL, FALSE); // Обновляем только нужную область
        }
        break;
    }

    case WM_DESTROY:

        if (hdcMem) {
            SelectObject(hdcMem, hbmOld);
            DeleteObject(hbmMem);
            DeleteDC(hdcMem);
        }
        PostQuitMessage(0);
        return 0;
    case WM_PAINT: {
        PAINTSTRUCT ps;
        HDC hdc = BeginPaint(hwnd, &ps);



        // Получаем размер клиентской области
        RECT clientRect;
        GetClientRect(hwnd, &clientRect);
        width = clientRect.right - clientRect.left;
        height = clientRect.bottom - clientRect.top;

        // Создаем контекст памяти для двойной буферизации
        HDC hdcMem = CreateCompatibleDC(hdc);
        HBITMAP hbmMem = CreateCompatibleBitmap(hdc, width, height);
        HGDIOBJ hOld = SelectObject(hdcMem, hbmMem);

        // Заполняем фон черным цветом
        HBRUSH blackBrush = (HBRUSH)GetStockObject(BLACK_BRUSH);
        FillRect(hdcMem, &clientRect, blackBrush);



        // Рисуем тетромино на контексте устройства в памяти
        COLORREF blueColor = RGB(0, 0, 255);
        DrawTetrominoRects(hdcMem, Figrnd, blueColor);
        for (int i = 0; i < Figrnd_out.size(); i++) {
            DrawTetrominoRects(hdcMem, Figrnd_out[i], blueColor);
        }

        // Создаем и выбираем перо
        HPEN hPen = CreatePen(PS_SOLID, 1, RGB(10, 10, 10));
        HPEN hOldPen = (HPEN)SelectObject(hdcMem, hPen);

        for (int i = 0; i < height / (scale * 11); i++) {
            // Задаем начальную позицию
            MoveToEx(hdcMem, i * scale * 11, 0, NULL); // Начальная точка
            // Рисуем линию до точки
            LineTo(hdcMem, i * scale * 11, height);
        }

        // Устанавливаем прозрачный фон для текста
        SetBkMode(hdcMem, TRANSPARENT);

        // Устанавливаем цвет текста в белый
        SetTextColor(hdcMem, RGB(255, 255, 255));

        // Создаем текст для отображения


        // Создаем шрифт
        HFONT hFont = CreateFont(
            26,                        // Высота шрифта
            0,                         // Средняя ширина символов
            0,                         // Угол наклона текста
            0,                         // Угол наклона базовой линии
            FW_NORMAL,                 // Толщина шрифта
            TRUE,                      // Курсив
            FALSE,                     // Подчеркнутый
            FALSE,                     // Зачеркнутый
            DEFAULT_CHARSET,           // Набор символов
            OUT_DEFAULT_PRECIS,        // Точность вывода
            CLIP_DEFAULT_PRECIS,       // Точность отсечения
            DEFAULT_QUALITY,           // Качество вывода
            FIXED_PITCH | FF_MODERN,   // Ширина и семейство шрифта
            TEXT("Monotype Corsiva")   // Имя шрифта
        );

        // Сохраняем старый шрифт, чтобы потом восстановить его
        HFONT hOldFont = (HFONT)SelectObject(hdcMem, hFont);

        std::string text = "Score " + std::to_string(Score);

        // Преобразуем std::string в std::wstring
        int size_needed = MultiByteToWideChar(CP_UTF8, 0, &text[0], (int)text.size(), NULL, 0);
        std::wstring wstrTo(size_needed, 0);
        MultiByteToWideChar(CP_UTF8, 0, &text[0], (int)text.size(), &wstrTo[0], size_needed);

        TextOut(hdcMem, 10, 10, text.c_str(), wstrTo.length());



        if (StopGame) {
            GameOver(hwnd);
            StopGame = false;
        }
        else {}


        // Восстанавливаем старый шрифт и удаляем созданный шрифт
        SelectObject(hdcMem, hOldFont);
        DeleteObject(hFont);

        // Восстанавливаем старое перо и удаляем созданное перо
        SelectObject(hdcMem, hOldPen);
        DeleteObject(hPen);

        // Копируем изображение из контекста устройства в памяти на экран
        BitBlt(hdc, 0, 0, clientRect.right - clientRect.left, clientRect.bottom - clientRect.top, hdcMem, 0, 0, SRCCOPY);

        // Освобождаем ресурсы
        SelectObject(hdcMem, hOld);
        DeleteObject(hbmMem);
        DeleteDC(hdcMem);

        EndPaint(hwnd, &ps);
        return 0;
    }

                 return 0;
    }
    return DefWindowProc(hwnd, uMsg, wParam, lParam);
}

int  window()
{
    LPCSTR CLASS_NAME = "Sample Window Class";

    // Получение дескриптора экземпляра
    HINSTANCE hInstance = GetModuleHandle(NULL);

    WNDCLASS wc = {};

    wc.lpfnWndProc = WindowProc;
    wc.hInstance = hInstance;
    wc.lpszClassName = CLASS_NAME;
    //wc.hbrBackground = (HBRUSH)(COLOR_WINDOW + 1);
    wc.hbrBackground = (HBRUSH)GetStockObject(BLACK_BRUSH); // Установка фона окна в черный цвет
    // Регистрация оконного класса
    RegisterClass(&wc);

    // Создание окна
    HWND hwnd = CreateWindowEx(
        0,
        CLASS_NAME,
        "Sample Window",
        WS_OVERLAPPEDWINDOW,
        // Размеры и положение окна
        CW_USEDEFAULT, CW_USEDEFAULT,  // Position
        676, 1062,                      // Размеры окна (ширина x высота)
        NULL,
        NULL,
        hInstance,
        NULL
    );

    if (hwnd == NULL) {
        return 0;
    }

    // Показ окна
    ShowWindow(hwnd, SW_SHOW);

    // Цикл обработки сообщений
    MSG msg = {};
    while (GetMessage(&msg, NULL, 0, 0)) {
        TranslateMessage(&msg);
        DispatchMessage(&msg);
        //  InvalidateRect(hwnd, NULL, TRUE);
    }
    return 0;
}


int main()
{
    Figs.push_back({ TetrominoFactory::CreateFig1(), TetrominoFactory::CreateFig1_1() });
    Figs.push_back({ TetrominoFactory::CreateFig2(), TetrominoFactory::CreateFig2_1(), TetrominoFactory::CreateFig2_2(), TetrominoFactory::CreateFig2_3() });
    Figs.push_back({ TetrominoFactory::CreateFig3(), TetrominoFactory::CreateFig3_1(), TetrominoFactory::CreateFig3_2(), TetrominoFactory::CreateFig3_3() });
    Figs.push_back({ TetrominoFactory::CreateFig4(), TetrominoFactory::CreateFig4_1(), TetrominoFactory::CreateFig4_2(), TetrominoFactory::CreateFig4_3() });
    Figs.push_back({ TetrominoFactory::CreateFig5() });
    Figs.push_back({ TetrominoFactory::CreateFig6(), TetrominoFactory::CreateFig6_1() });
    Figs.push_back({ TetrominoFactory::CreateFig7(), TetrominoFactory::CreateFig7_1() });

    Scale(scale, Figs);

    Startgame();

    window();


    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
